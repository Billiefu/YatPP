#include <iostream>
#include <hip/hip_runtime.h>

// CUDA kernel
__global__ void helloWorldKernel() {
    int blockId = blockIdx.x;
    int threadRow = threadIdx.y;
    int threadCol = threadIdx.x;
    
    printf("Hello World from Thread (%d, %d) in Block %d!\n", threadRow, threadCol, blockId);
}

int main() {
    int n, m, k;

    // 从标准输入读取参数
    std::cout << "Enter n (blocks), m (threads per block in y), k (threads per block in x): ";
    std::cin >> n >> m >> k;

    if (n < 1 || n > 32 || m < 1 || m > 32 || k < 1 || k > 32) {
        std::cerr << "Error: All values must be between 1 and 32." << std::endl;
        return 1;
    }

    std::cout << "Hello World from the host!" << std::endl;

    dim3 blockDim(k, m);  // x=k, y=m
    dim3 gridDim(n);      // 1D grid with n blocks

    helloWorldKernel<<<gridDim, blockDim>>>();
    hipDeviceSynchronize();  // 等待GPU执行完毕

    return 0;
}
