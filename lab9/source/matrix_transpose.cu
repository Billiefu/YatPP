#include <iostream>
#include <cstdlib>
#include <hip/hip_runtime.h>

#define BLOCK_SIZE 16  // 可调，尝试 16、32 等不同大小

// CUDA 核函数：进行矩阵转置
__global__ void transpose(float* A, float* A_T, int n) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < n && col < n) {
        A_T[col * n + row] = A[row * n + col];
    }
}

// 初始化矩阵
void generateMatrix(float* mat, int n) {
    for (int i = 0; i < n * n; ++i) {
        mat[i] = static_cast<float>(rand() % 100);
    }
}

// 打印矩阵
void printMatrix(const float* mat, int n) {
    for (int i = 0; i < n; ++i) {
        for (int j = 0; j < n; ++j)
            std::cout << mat[i * n + j] << "\t";
        std::cout << "\n";
    }
}

int main(int argc, char* argv[]) {
    if (argc != 2) {
        std::cerr << "Usage: ./matrix_transpose <matrix_size>\n";
        return 1;
    }

    int n = atoi(argv[1]);
    size_t size = n * n * sizeof(float);

    // 分配主机内存
    float *h_A = (float*)malloc(size);
    float *h_A_T = (float*)malloc(size);

    generateMatrix(h_A, n);

    // 分配设备内存
    float *d_A, *d_A_T;
    hipMalloc(&d_A, size);
    hipMalloc(&d_A_T, size);

    // 复制数据到设备
    hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);

    // 设置 CUDA 定时器
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // 启动核函数
    dim3 blockSize(BLOCK_SIZE, BLOCK_SIZE);
    dim3 gridSize((n + BLOCK_SIZE - 1) / BLOCK_SIZE, (n + BLOCK_SIZE - 1) / BLOCK_SIZE);

    hipEventRecord(start);
    transpose<<<gridSize, blockSize>>>(d_A, d_A_T, n);
    hipEventRecord(stop);

    // 复制结果回主机
    hipMemcpy(h_A_T, d_A_T, size, hipMemcpyDeviceToHost);

    // 计算耗时
    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);

    // 打印结果
    // std::cout << "Matrix A:\n";
    // printMatrix(h_A, n);

    // std::cout << "\nTranspose of A (A^T):\n";
    // printMatrix(h_A_T, n);

    std::cout << "\nTime taken for CUDA transpose: " << milliseconds << " ms\n";

    // 清理
    free(h_A);
    free(h_A_T);
    hipFree(d_A);
    hipFree(d_A_T);

    return 0;
}
